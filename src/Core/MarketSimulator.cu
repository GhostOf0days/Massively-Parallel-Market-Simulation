#include "hip/hip_runtime.h"
#include "MarketSimulator.h"
#include "../Utils/CSVReader.h"
#include "../Utils/CSVWriter.h"
#include "../Utils/Logger.h"
#include "../Strategies/TrendFollowing.h"
#include "../Strategies/MeanReversion.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void simulateMarketKernel(Asset* assets, int numAssets, hiprandState* randStates) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numAssets) {
        hiprandState localState = randStates[idx];
        double randPrice = hiprand_normal_double(&localState) * 0.01 + assets[idx].getPrice();
        double randVolume = hiprand_uniform_double(&localState) * 1000.0;
        assets[idx].setPrice(randPrice);
        assets[idx].setVolume(randVolume);
        randStates[idx] = localState;
    }
}

MarketSimulator::MarketSimulator(int numAssets, int numTraders, int numTimeSteps)
    : numAssets(numAssets), numTraders(numTraders), numTimeSteps(numTimeSteps) {
    // Initialize assets
    CSVReader assetReader("data/assets.csv");
    assets = assetReader.readAssets();

    // Initialize traders
    CSVReader traderReader("data/traders.csv");
    traders = traderReader.readTraders();

    // Initialize strategies
    strategies.push_back(new TrendFollowing());
    strategies.push_back(new MeanReversion());
}

void MarketSimulator::run() {
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    // Allocate memory on GPU
    Asset* d_assets;
    hipMalloc(&d_assets, numAssets * sizeof(Asset));
    hiprandState* d_randStates;
    hipMalloc(&d_randStates, numAssets * sizeof(hiprandState));

    // Initialize random states for CUDA
    hiprandState* h_randStates = new hiprandState[numAssets];
    for (int i = 0; i < numAssets; i++) {
        hiprand_init(rank * numAssets + i, 0, 0, &h_randStates[i]);
    }
    hipMemcpy(d_randStates, h_randStates, numAssets * sizeof(hiprandState), hipMemcpyHostToDevice);
    delete[] h_randStates;

    for (int t = 0; t < numTimeSteps; t++) {
        // Simulate market on GPU
        hipMemcpy(d_assets, assets.data(), numAssets * sizeof(Asset), hipMemcpyHostToDevice);
        int blockSize = 256;
        int numBlocks = (numAssets + blockSize - 1) / blockSize;
        simulateMarketKernel<<<numBlocks, blockSize>>>(d_assets, numAssets, d_randStates);
        hipMemcpy(assets.data(), d_assets, numAssets * sizeof(Asset), hipMemcpyDeviceToHost);

        // Execute trading strategies
        for (auto strategy : strategies) {
            strategy->execute(assets, traders);
        }

        // Gather market data and trading orders from all ranks
        MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, assets.data(), numAssets * sizeof(Asset), MPI_BYTE, MPI_COMM_WORLD);
        MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, traders.data(), numTraders * sizeof(Trader), MPI_BYTE, MPI_COMM_WORLD);

        // Write market snapshot to CSV file
        if (rank == 0) {
            CSVWriter snapshotWriter("results/snapshot_" + std::to_string(t) + ".csv");
            snapshotWriter.writeAssets(assets);
            snapshotWriter.writeTraders(traders);
            Logger::log("Snapshot written for time step " + std::to_string(t));
        }
    }

    // Cleanup
    hipFree(d_assets);
    hipFree(d_randStates);
    for (auto strategy : strategies) {
        delete strategy;
    }
}